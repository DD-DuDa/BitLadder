// Copyright (c) 2023, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "../flash_fwd_launch_template.h"

// template<>
// void run_kvcache_qpack_<cutlass::half_t, 128, 1, 4, 32>(Flash_fwd_params &params, hipStream_t stream) {
//     run_kvcache_qpack_hdim128<cutlass::half_t, 1, 4, 32>(params, stream);
// }
// template<>
// void run_kvcache_qpack_<cutlass::half_t, 128, 1, 4, 64>(Flash_fwd_params &params, hipStream_t stream) {
//     run_kvcache_qpack_hdim128<cutlass::half_t, 1, 4, 64>(params, stream);
// }
template<>
void run_kvcache_qpack_<cutlass::half_t, 128, 1, 4, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_kvcache_qpack_hdim128<cutlass::half_t, 1, 4, 128>(params, stream);
}


// template<>
// void run_kvcache_qpack_<cutlass::half_t, 128, 0, 4, 32>(Flash_fwd_params &params, hipStream_t stream) {
//     run_kvcache_qpack_hdim128<cutlass::half_t, 0, 4, 32>(params, stream);
// }
// template<>
// void run_kvcache_qpack_<cutlass::half_t, 128, 0, 4, 64>(Flash_fwd_params &params, hipStream_t stream) {
//     run_kvcache_qpack_hdim128<cutlass::half_t, 0, 4, 64>(params, stream);
// }
// template<>
// void run_kvcache_qpack_<cutlass::half_t, 128, 0, 4, 128>(Flash_fwd_params &params, hipStream_t stream) {
//     run_kvcache_qpack_hdim128<cutlass::half_t, 0, 4, 128>(params, stream);
// }


